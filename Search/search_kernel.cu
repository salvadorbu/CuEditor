#include "hip/hip_runtime.h"
#include "search_kernel.cuh"
#include <hip/device_functions.h>

__global__ void searchKernel(const char* text, size_t textLength,
    const char* pattern, size_t patternLength,
    unsigned long long* results, size_t* numResults,
    size_t totalResults,
    size_t offset)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (size_t i = idx; i < textLength - patternLength + 1; i += stride) {
        bool match = true;
        for (size_t j = 0; j < patternLength; j++) {
            if (text[i + j] != pattern[j]) {
                match = false;
                break;
            }
        }

        if (match) {
            size_t resultIdx = atomicAdd(numResults, 1);
            if (resultIdx < totalResults) {
                results[resultIdx] = i + offset;
            }
        }
    }
}

extern "C" bool LaunchSearchKernel(
    const char* d_fileData, size_t chunkSize,
    const char* d_pattern, size_t patternLength,
    unsigned long long* d_results, size_t totalResults,
    size_t * d_numResults,
    size_t chunkOffset
) {
    int threadsPerBlock = 256;
    int blocks = (int)((chunkSize + threadsPerBlock - 1) / threadsPerBlock);

    size_t zero = 0;
    hipError_t cudaStatus = hipMemcpy(d_numResults, &zero, sizeof(size_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        return false;
    }

    searchKernel << <blocks, threadsPerBlock >> > (
        d_fileData, chunkSize,
        d_pattern, patternLength,
        d_results, d_numResults,
        totalResults,
        chunkOffset
        );

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        return false;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        return false;
    }

    return true;
}
